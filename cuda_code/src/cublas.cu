#include <hipblas.h>
#include <cstdio>
#include <string>
#include <fstream>

#define CHECK_CUBLAS(Expr) { \
    int err = (Expr); \
    if (err != 0) { \
        printf("cuBLAS error %d at line %d\n", err, __LINE__); \
    } \
}

void gemm(hipblasHandle_t handle,
          int m,
          int n,
          int k,
          const void *alpha,
          const void *beta,
          hipDataType input_type,
          const void *A,
          const void *B,
          hipDataType output_type,
          void *C,
#if __CUDACC_VER_MAJOR__ >= 11
          hipblasComputeType_t compute_type,
#else
          hipDataType compute_type,
#endif
          hipblasGemmAlgo_t algo) {
    CHECK_CUBLAS(hipblasGemmEx(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
        alpha, B, input_type, n, A, input_type, k,
        beta, C, output_type, n, compute_type, algo));
}

int main(int argc, char **argv) {
    int m, n, k = 0;

    if(argc != 4) {
        printf("Usage: %s <m> <n> <k>\n", argv[0]);
        return 1;
    }
    else {
        m = atoi(argv[1]);
        n = atoi(argv[2]);
        k = atoi(argv[3]);
        // gpu_runtime::load_graph(__host_edges, argv[4], false);

        printf("m = %d, n = %d, k = %d\n", m, n, k);
    }

    // int m = 403394;
    // int n = 16;
    // int k = 22;

    // int m = 5120;
    // int n = 4096;
    // int k = 4096;

    float alpha = 1;
    float beta = 0;

    hipDataType input_type = HIP_R_32F;
    hipDataType output_type = HIP_R_32F;
#if __CUDACC_VER_MAJOR__ >= 11
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;
#else
    hipDataType compute_type = HIP_R_32F;
#endif
    hipblasGemmAlgo_t algo = HIPBLAS_GEMM_DEFAULT;

    int iter = 200;

    void *A, *B, *C;
    hipMalloc(&A, m * k * sizeof(float));
    hipMalloc(&B, k * n * sizeof(float));
    hipMalloc(&C, m * n * sizeof(float));

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // warmup
    gemm(handle, m, n, k, &alpha, &beta, input_type, A, B,
         output_type, C, compute_type, algo);

    hipEventRecord(start);
    for (int i = 0; i < iter; ++i) {
        gemm(handle, m, n, k, &alpha, &beta, input_type, A, B,
             output_type, C, compute_type, algo);
    }
    hipEventRecord(stop);

    float time_ms = 0.f;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_ms, start, stop);

    long ops = (long)m * n * k * 2;
    double gops = ((double)ops / 1e9) / ((double)time_ms / iter / 1e3);
    printf("%f Gops\n", gops);
    
    printf("Time: %fms\n", time_ms/iter); 
    
    std::fstream fp;
	fp.open(rst_file, std::ios::out|std::ios::app);
	fp << time_ms/iter << "," << gops << std::endl;
	fp.close();

    hipFree(A);
    hipFree(B);
    hipFree(C);

}


